#include "hip/hip_runtime.h"
/*
------------------DR VASILIOS KELEFOURAS-----------------------------------------------------
------------------COMP3001 ------------------------------------------------------------------
------------------PARALLEL PROGAMMING MODULE-------------------------------------------------
------------------UNIVERSITY OF PLYMOUTH, SCHOOL OF ENGINEERING, COMPUTING AND MATHEMATICS---
*/


#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

int compare();

#define N 1024 //input size
#define TIMES_TO_RUN 10000 //this is used in case the kernel runs for a very small perior of time
#define TILE 32

float A[N][N], Atranspose[N][N]; //our arrays


#define MAX_NUMBER_OF_BLOCKS_PER_DIM 65535 //max number of blocks that our GPU can handle (for one dimension only)

__device__ float device_A[N][N]; //allocate the device arrays statically (global GPU memory)
__device__ float device_Atranspose[N][N]; //allocate the device arrays statically (global GPU memory)


void initialize() {//this routine initizalies the array with some values

	int i, j;
	for (i = 0; i < N; i++)
		for (j = 0; j < N; j++) {
			A[i][j] = (float)(((i * 3 + j) % 1000) + 0.01);
		}
}



//in this implementation, each thread copies a single element
__global__ void transpose_ver1() {

	int i = blockIdx.x * blockDim.x + threadIdx.x; //2d grid, 2d blocks

	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < N && j < N) { //this is not necessary, but good practice
		device_Atranspose[i][j] = device_A[j][i]; //each thread copies one element
	}

}

//in this implementation, each thread copies multiple elements
__global__ void transpose_ver4() {

	int x = blockIdx.x * TILE + threadIdx.x;//2d grid, 2d blocks
	int y = blockIdx.y * TILE + threadIdx.y;

	for (int m = 0; m < TILE; m += 8) {
		device_Atranspose[x][y + m] = device_A[y + m][x];//each thread copies more than one elements
	}

}

//this is like implementation_ver4 but shared memory is used
__global__ void transpose_ver5() {

	__shared__ float tile[TILE][TILE]; //try to use 'float tile[TILE][TILE+1];'

	int x = blockIdx.x * TILE + threadIdx.x;//2d grid, 2d blocks
	int y = blockIdx.y * TILE + threadIdx.y;

	for (int m = 0; m < TILE; m += 8) {
		tile[threadIdx.y + m][threadIdx.x] = device_A[y + m][x];//each thread copies more than one elements
	}

	__syncthreads(); //all the threads wait here until the tile array has been initialized

	x = blockIdx.y * TILE + threadIdx.x;//transpose block offset
	y = blockIdx.x * TILE + threadIdx.y;

	for (int m = 0; m < TILE; m += 8) {
		device_Atranspose[y + m][x] = tile[threadIdx.x][threadIdx.y + m];
	}

}


//this kernel does a normal copy (not transpose) from A to Atranspose. Atranspose it NOT the transpose of A here.
//the purpose of this kernel is to measure the maximum performance of copying two matrices
//143 GB/sec
__global__ void normal_copy() {

	int x = blockIdx.x * TILE + threadIdx.x; //2d grid, 2d blocks
	int y = blockIdx.y * TILE + threadIdx.y;

	for (int m = 0; m < TILE; m += 8) {
		device_Atranspose[y + m][x] = device_A[y + m][x];//each thread copies more than one elements
	}

}





int main(int argc, char* argv[])
{
	hipError_t cudaStatus;
	initialize();

	//create the cuda timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;
	int devId = 0;
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devId);
	printf("\n Device: %s \n", prop.name);

	/* Copy the A array from the HOST memory to the DEVICE memory */
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(device_A), A, N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		return -1;
	}


	//parameters for normal_copy() 
	//parameters for transpose_ver4()
	//parameters for transpose_ver5()
	dim3 dimBlock(TILE, 8, 1);
	dim3 dimGrid((N + TILE - 1) / TILE, (N + TILE - 1) / TILE, 1);

	//parameters for transpose_ver1()
		//dim3 dimBlock(TILE, TILE, 1);
		//dim3 dimGrid((N + TILE - 1) / TILE, (N + TILE - 1) / TILE, 1);

	hipEventRecord(start, 0);
	
//in CUDA you do not have to run the program many times to get an accurate ex.time. However, the first time you run a kernel it normally takes more. So you need to include one - two extra runs (this is also known as warm up time). The CUDA timers have an accuracy of 1 msec.
	for (int it = 0; it < TIMES_TO_RUN; it++) {
		//normal_copy << <dimGrid, dimBlock >> > ();
		transpose_ver5 << <dimGrid, dimBlock >> > ();
	//	transpose_ver1 << <dimGrid, dimBlock >> > ();
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	double bandwidth = double((double)(2 * N * N * sizeof(float))) / ((elapsed_time / 1000) / TIMES_TO_RUN);
	printf("\nElapsed time in msecs = %f - GB/sec = %f", elapsed_time, bandwidth / 1000000000);


	//dim3 dimBlock(TILE, TILE, 1);
	//dim3 dimGrid((N + TILE - 1) / TILE, (N + TILE - 1) / TILE, 1);
	//transpose_ver2 << <dimGrid, dimBlock >> > ();
	//transpose_ver2 << <dimGrid, dimBlock >> > ();
	//transpose_ver3 << <dimGrid, dimBlock >> > ();


	//transpose_ver4 << <dimGrid, dimBlock >> > ();




	hipEventDestroy(start);
	hipEventDestroy(stop);

	/*  Handling function of the CUDA runtime application programming interface.
	*   Returns the last error from a runtime call.
	*/
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(error));
	}

	/* Copy back the result from the DEVICE memory to the HOST memory */
	cudaStatus = hipMemcpyFromSymbol(Atranspose, HIP_SYMBOL(device_Atranspose), N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		return -1;
	}



	compare();


	/* Destroy all allocations and reset all state on the current device in the current process */
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		printf("\ncuda Reset failed!");
		return -1;
	}

	return 0;
}

int compare() {

	int i, j;
	for (i = 0; i < N; i++)
		for (j = 0; j < N; j++) {
			if (fabs( (A[i][j] - Atranspose[j][i]) / A[i][j]) > 0.00001) {
				printf("\n\wrong results at (%d, %d); they are (%f, %f)\n", i, j,A[i][j],Atranspose[j][i]);
				return -1;
			}
		}
	printf("\nResults are correct\n");
	return 0;
}

