#include "hip/hip_runtime.h"
/*
------------------DR VASILIOS KELEFOURAS-----------------------------------------------------
------------------COMP3001 ------------------------------------------------------------------
------------------PARALLEL PROGAMMING MODULE-------------------------------------------------
------------------UNIVERSITY OF PLYMOUTH, SCHOOL OF ENGINEERING, COMPUTING AND MATHEMATICS---
*/

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <>
#include <stdlib.h>
#include <stdio.h>

int compare(const int* a, const int* b, const int* c);

#define VECTOR_LENGTH 10000 //input size


#define MAX_NUMBER_OF_BLOCKS 65535 //max number of blocks that our GPU can handle (for one dimension only)

/*
* Function:  random_ints
* --------------------
*  generates a 1D vector of size equal to num_elements where each element is between 1 and 100
*
*  Input:    num_elements - int - number of elements composing the 1D vector
*
*  Output:   x - int* (pointer to int) - 1D vector of num_elements elements
*/
void random_ints(int* x, int num_elements) {
	int i;
	int min = 1;
	int range = 100;
	for (i = 0; i < num_elements; i++)
	{
		x[i] = rand() % range + 1;
	}
}

/*
* Kernel Function:  addWithBlocks
* --------------------
*  computes the element-wise sum of two 1D vectors of size num_elements by using the GPU
*  the kernel uses a one-dimensional grid of a one-dimensional block
*  the one-dimensional grid is composed of MAX_NUMBER_OF_BLOCKS number of blocks
*  the one-dimensional block is composed of only one single thread (parallelism of blocks)
*
*  Input:    a -  int* (pointer to int) - first 1D vector of num_elements elements
*            b -  int* (pointer to int) - second 1D vector of num_elements elements
*
*  Output:   c - int* (pointer to int) - 1D vector resulting from the element-wise sum of a and b vectors
*/
__global__ void addWithBlocks(int* a, int* b, int* c) {
	/* the index of the block in the 1D grid along the x-dimension is used to access to the elements of the array */
	if (blockIdx.x < VECTOR_LENGTH) {

		c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];

	}
	//WHAT IF VECTOR LENGTH IS LARGER THAN 65535?
}


/* In C, the "main" function is treated the same as every function,
*  it has a return type (and in some cases accepts inputs via parameters).
*  The only difference is that the main function is "called" by the operating
*  system when the user runs the program.
*  Thus the main function is always the first code executed when a program starts.
*  This function returns an integer representing the application software status.
*/
int main(int argc, char* argv[])
{
	hipError_t cudaStatus;

	/* Pointer to an integer representing the first input 1D vector on the memory of the HOST */
	int* host_a;
	/* Dynamic allocation of the amount of space in the memory of the host needed to store the values of the elements of the first input 1D vector */
	host_a = (int*)malloc(VECTOR_LENGTH * sizeof(int));
	if (host_a == NULL) {
		printf("\n memory not allocated");
		return -1;
	}

	/* Fill the elements of the first 1D vector stored in the HOST memory using random_ints function */
	random_ints(host_a, VECTOR_LENGTH);

	/* Pointer to an integer representing the second input 1D vector on the memory of the HOST */
	int* host_b;
	/* Dynamic allocation of the amount of space in the memory of the host needed to store the values of the elements of the second input 1D vector */
	host_b = (int*)malloc(VECTOR_LENGTH * sizeof(int));
	if (host_b == NULL) {
		printf("\n memory not allocated");
		free(host_a);
		return -1;
	}

	/* Fill the elements of the second 1D vector stored in the HOST memory using random_ints function */
	random_ints(host_b, VECTOR_LENGTH);

	/* Pointer to an integer representing the output 1D vector on the memory of the HOST resulting from the element-wise sum of the first and the second 1D vectors */
	int* host_c;
	host_c = (int*)malloc(VECTOR_LENGTH * sizeof(int));
	if (host_c == NULL) {
		printf("\n memory not allocated");
		free(host_a); free(host_b);
		return -1;
	}

	/* Pointer to an integer representing the first input 1D vector on the memory of the DEVICE */
	int* device_a;
	/* Dynamic allocation of the amount of space in the memory of the device needed to store the values of the elements of the second input 1D vector */
	cudaStatus = hipMalloc((void**)&device_a, VECTOR_LENGTH * sizeof(int));
	if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available 
		printf("\ncudaMalloc failed!");
		free(host_a); free(host_b); free(host_c);
		return -1;
	}

	/* Pointer to an integer representing the second input 1D vector on the memory of the DEVICE */
	int* device_b;
	/* Dynamic allocation of the amount of space in the memory of the device needed to store the values of the elements of the second input 1D vector */
	cudaStatus = hipMalloc((void**)&device_b, VECTOR_LENGTH * sizeof(int));
	if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available 
		printf("\ncudaMalloc failed!");
		free(host_a); free(host_b); free(host_c); hipFree(device_a);
		return -1;
	}

	/* Pointer to an integer representing the output 1D vector on the memory of the DEVICE resulting from the element-wise sum of the first and the second 1D vectors */
	int* device_c;
	/* Dynamic allocation of the amount of space in the memory of the device needed to store the values of the elements of the second input 1D vector */
	cudaStatus = hipMalloc((void**)&device_c, VECTOR_LENGTH * sizeof(int));
	if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available 
		printf("\ncudaMalloc failed!");
		free(host_a); free(host_b); free(host_c); hipFree(device_a); hipFree(device_b);
		return -1;
	}

	/* Copy the first 1D vector from the HOST memory to the DEVICE memory */
	cudaStatus = hipMemcpy(device_a, host_a, VECTOR_LENGTH * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		free(host_a); free(host_b); free(host_c); hipFree(device_a); hipFree(device_b); hipFree(device_c);
		return -1;
	}

	/* Copy the second 1D vector from the HOST memory to the DEVICE memory */
	cudaStatus = hipMemcpy(device_b, host_b, VECTOR_LENGTH * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		free(host_a); free(host_b); free(host_c); hipFree(device_a); hipFree(device_b); hipFree(device_c);
		return -1;
	}

	/* Definition of the structure of the execution configuration of the kernel based on parallelism of blocks:
	*  - The 1D grid is composed of MAX_NUMBER_OF_BLOCKS amount of blocks in the x-dimension of the grid
	*  - Each 1D block is composed of only one single thread
	*/
	dim3 dimGrid(MAX_NUMBER_OF_BLOCKS, 1, 1);
	dim3 dimBlock(1, 1, 1);

	/* Invocation of the kernel addWithBlocks with the execution configuration previously defined */
	addWithBlocks << <dimGrid, dimBlock >> > (device_a, device_b, device_c);

	/*  Handling function of the CUDA runtime application programming interface.
	*   Returns the last error from a runtime call.
	*/
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(error));
	}

	/* Copy back the result of the element-wise sum of the first and the second 1D vectors computed by the DEVICE from the DEVICE memory to the HOST memory */
	cudaStatus = hipMemcpy(host_c, device_c, VECTOR_LENGTH * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		free(host_a); free(host_b); free(host_c); hipFree(device_a); hipFree(device_b); hipFree(device_c);
		return -1;
	}

	compare(host_a, host_b, host_c);


	/* Deallocation of the HOST memory previously allocated by malloc storing the first 1D vector */
	free(host_a);
	/* Deallocation of the HOST memory previously allocated by malloc storing the second 1D vector */
	free(host_b);
	/* Deallocation of the HOST memory previously allocated by malloc storing the output 1D vector */
	free(host_c);
	/* Deallocation of the DEVICE memory previously allocated by hipMalloc storing the first 1D vector */
	hipFree(device_a);
	/* Deallocation of the DEVICE memory previously allocated by hipMalloc storing the second 1D vector */
	hipFree(device_b);
	/* Deallocation of the DEVICE memory previously allocated by hipMalloc storing the output 1D vector */
	hipFree(device_c);

	/* Destroy all allocations and reset all state on the current device in the current process */
	hipDeviceReset();

	return 0;
}

int compare(const int* a, const int* b, const int* c) {

	int i;
	for (i = 0; i < VECTOR_LENGTH; i++) {
		if ((a[i] + b[i]) != c[i]) {
			printf("\n\wrong results\n");
			return -1;
		}
	}
	printf("\nResults are correct\n");
	return 0;
}
